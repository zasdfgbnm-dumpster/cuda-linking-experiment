
#include <hip/hip_runtime.h>
template<long long i>
__global__ void add_kernel(float *a, float *b, float *c, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        c[idx] = a[idx] + b[idx];
    }
}

template<long long l = 0, long long r = 1000>
void _add(float *a, float *b, float *c, int N) {
    if constexpr (l == r) {
        add_kernel<l><<<(N + 255) / 256, 256>>>(a, b, c, N);
    } else if constexpr (l + 1 == r) {
        add_kernel<l><<<(N + 255) / 256, 256>>>(a, b, c, N);
        add_kernel<r><<<(N + 255) / 256, 256>>>(a, b, c, N);
    } else {
        constexpr long long m = (l + r) / 2;
        _add<l, m>(a, b, c, N);
        _add<m, r>(a, b, c, N);
    }
}

void add(float *a, float *b, float *c, int N) {
    _add(a, b, c, N);
}
